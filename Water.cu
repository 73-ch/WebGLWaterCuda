#include "hip/hip_runtime.h"
//
// Created by nami on 11/13/20.
//


#include "Water.cuh"
#include "hip/hip_runtime.h"
#include "math_functions.h"
#include "hip/hip_vector_types.h"

extern "C" {

void setupCuda() {

}
void setupFbo() {

}


void stepSimulation() {

    dim3 block(16,16,1);
    dim3 grid(width / block.x, height / block.y, 1);

    updateWater();
}

void updateNormals() {

}

}


__global__ void updateWater(float4 *g_data, float4 *g_odata, uint2 size, int2 delta, float attenuation) {
    uint gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    uint gtidy = blockIdx.y * blockDim.y + threadIdx.y;

    float4 data = g_data[gtidx + gtidy * size.x];

    float average = g_data[gtidx + delta.x + gtidy * size.x].x;
    average += g_data[gtidx - delta.x + gtidy * size.x].x;
    average += g_data[gtidx + (gtidy + delta.y) * size.x].x;
    average += g_data[gtidx + (gtidy - delta.y) * size.x].x;
    average *= 0.25;

    data.y += (average - data.x) * 2.0;
    data.y *= attenuation;

    data.x += data.y;

    g_odata[gtidx + gtidy * size.x] = data;
}